#include "hip/hip_runtime.h"
#include <stdio.h>

#include "correlation_cuda_kernel.cuh"

#define CUDA_NUM_THREADS 1024
#define THREADS_PER_BLOCK 32
#define FULL_MASK 0xffffffff

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

using at::Half;

template<typename scalar_t>
__forceinline__ __device__ scalar_t warpReduceSum(scalar_t val) {
        for (int offset = 16; offset > 0; offset /= 2)
                val += __shfl_down_sync(FULL_MASK, val, offset);
        return val;
}

template<typename scalar_t>
__forceinline__ __device__ scalar_t blockReduceSum(scalar_t val) {

        static __shared__ scalar_t shared[32];
        int lane = threadIdx.x % warpSize;
        int wid = threadIdx.x / warpSize;

        val = warpReduceSum(val);

        if (lane == 0)
                shared[wid] = val;

        __syncthreads();

        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

        if (wid == 0)
                val = warpReduceSum(val);

        return val;
}


template <typename scalar_t>
__global__ void channels_first(const scalar_t* __restrict__ input, scalar_t* rinput, int channels, int height, int width, int pad_size)
{

    // n (batch size), c (num of channels), y (height), x (width)
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;

    int ch_off = threadIdx.x;
    scalar_t value;

    int dimcyx = channels * height * width;
    int dimyx = height * width;

    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
      value = input[n * dimcyx + c * dimyx + y * width + x];
      rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = value;
    }
}


template<typename scalar_t>
__global__ void correlation_forward(scalar_t* __restrict__ output, const int nOutputChannels,
                const int outputHeight, const int outputWidth, const scalar_t* __restrict__ rInput1,
                const int nInputChannels, const int inputHeight, const int inputWidth,
                const scalar_t* __restrict__ rInput2, const int pad_size, const int kernel_size,
                const int max_displacement, const int stride1, const int stride2) {

        int32_t pInputWidth = inputWidth + 2 * pad_size;
        int32_t pInputHeight = inputHeight + 2 * pad_size;

        int32_t kernel_rad = (kernel_size - 1) / 2;

        int32_t displacement_rad = max_displacement / stride2;

        int32_t displacement_size = 2 * displacement_rad + 1;

        int32_t n = blockIdx.x;
        int32_t y1 = blockIdx.y * stride1 + max_displacement;
        int32_t x1 = blockIdx.z * stride1 + max_displacement;
        int32_t c = threadIdx.x;

        int32_t pdimyxc = pInputHeight * pInputWidth * nInputChannels;

        int32_t pdimxc = pInputWidth * nInputChannels;

        int32_t pdimc = nInputChannels;

        int32_t tdimcyx = nOutputChannels * outputHeight * outputWidth;
        int32_t tdimyx = outputHeight * outputWidth;
        int32_t tdimx = outputWidth;

        int32_t nelems = kernel_size * kernel_size * pdimc;

        // element-wise product along channel axis
        for (int tj = -displacement_rad; tj <= displacement_rad; ++tj) {
                for (int ti = -displacement_rad; ti <= displacement_rad; ++ti) {
                        int x2 = x1 + ti * stride2;
                        int y2 = y1 + tj * stride2;

                        float acc0 = 0.0f;

                        for (int j = -kernel_rad; j <= kernel_rad; ++j) {
                                for (int i = -kernel_rad; i <= kernel_rad; ++i) {
                                        // THREADS_PER_BLOCK
                                        #pragma unroll
                                        for (int ch = c; ch < pdimc; ch += blockDim.x) {

                                                int indx1 = n * pdimyxc + (y1 + j) * pdimxc
                                                                + (x1 + i) * pdimc + ch;
                                                int indx2 = n * pdimyxc + (y2 + j) * pdimxc
                                                                + (x2 + i) * pdimc + ch;
                                                acc0 += static_cast<float>(rInput1[indx1] * rInput2[indx2]);
                                        }
                                }
                        }

                        if (blockDim.x == warpSize) {
                            __syncwarp();
                            acc0 = warpReduceSum(acc0);
                        } else {
                            __syncthreads();
                            acc0 = blockReduceSum(acc0);
                        }

                        if (threadIdx.x == 0) {

                                int tc = (tj + displacement_rad) * displacement_size
                                                + (ti + displacement_rad);
                                const int tindx = n * tdimcyx + tc * tdimyx + blockIdx.y * tdimx
                                                + blockIdx.z;
                                output[tindx] = static_cast<scalar_t>(acc0 / nelems);
                        }
            }
        }
}


template <typename scalar_t>
__global__ void correlation_backward_input1(int item, scalar_t* gradInput1, int nInputChannels, int inputHeight, int inputWidth, 
                                            const scalar_t* __restrict__ gradOutput, int nOutputChannels, int outputHeight, int outputWidth, 
                                            const scalar_t* __restrict__ rInput2, 
                                            int pad_size,
                                            int kernel_size,
                                            int max_displacement,
                                            int stride1,
                                            int stride2)
  {
    // n (batch size), c (num of channels), y (height), x (width)

    int n = item; 
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;
    int tch_off = threadIdx.x;

    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;

    int xmin = (x - kernel_rad - max_displacement) / stride1;
    int ymin = (y - kernel_rad - max_displacement) / stride1;

    int xmax = (x + kernel_rad - max_displacement) / stride1;
    int ymax = (y + kernel_rad - max_displacement) / stride1;

    if (xmax < 0 || ymax < 0 || xmin >= outputWidth || ymin >= outputHeight) {
        // assumes gradInput1 is pre-allocated and zero filled
      return;
    }

    if (xmin > xmax || ymin > ymax) {
        // assumes gradInput1 is pre-allocated and zero filled
        return;
    }

    xmin = max(0,xmin);
    xmax = min(outputWidth-1,xmax);

    ymin = max(0,ymin);
    ymax = min(outputHeight-1,ymax);

    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;

    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;

    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    int odimcyx = nInputChannels * inputHeight* inputWidth;
    int odimyx = inputHeight * inputWidth;
    int odimx = inputWidth;

    scalar_t nelems = kernel_size * kernel_size * nInputChannels;

    __shared__ scalar_t prod_sum[THREADS_PER_BLOCK];
    prod_sum[tch_off] = 0;

    for (int tc = tch_off; tc < nOutputChannels; tc += THREADS_PER_BLOCK) {

      int i2 = (tc % displacement_size - displacement_rad) * stride2;
      int j2 = (tc / displacement_size - displacement_rad) * stride2;

      int indx2 = n * pdimyxc + (y + j2)* pdimxc + (x + i2) * pdimc + c;
      
      scalar_t val2 = rInput2[indx2];

      for (int j = ymin; j <= ymax; ++j) {
        for (int i = xmin; i <= xmax; ++i) {
          int tindx = n * tdimcyx + tc * tdimyx + j * tdimx + i;
          prod_sum[tch_off] += gradOutput[tindx] * val2;
        }
      }
    }
    __syncthreads();

    if(tch_off == 0) {
      scalar_t reduce_sum = 0;
      for(int idx = 0; idx < THREADS_PER_BLOCK; idx++) {
          reduce_sum += prod_sum[idx];
      }
      const int indx1 = n * odimcyx + c * odimyx + (y - pad_size) * odimx + (x - pad_size);
      gradInput1[indx1] = reduce_sum / nelems;
    }

}

template <typename scalar_t>
__global__ void correlation_backward_input2(int item, scalar_t*  gradInput2, int nInputChannels, int inputHeight, int inputWidth,
                                            const scalar_t* __restrict__ gradOutput, int nOutputChannels, int outputHeight, int outputWidth,
                                            const scalar_t* __restrict__ rInput1,
                                            int pad_size,
                                            int kernel_size,
                                            int max_displacement,
                                            int stride1,
                                            int stride2)
{
    // n (batch size), c (num of channels), y (height), x (width)

    int n = item;
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;

    int tch_off = threadIdx.x;

    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;

    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;

    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;

    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    int odimcyx = nInputChannels * inputHeight* inputWidth;
    int odimyx = inputHeight * inputWidth;
    int odimx = inputWidth;

    scalar_t nelems = kernel_size * kernel_size * nInputChannels;

    __shared__ scalar_t prod_sum[THREADS_PER_BLOCK];
    prod_sum[tch_off] = 0;

    for (int tc = tch_off; tc < nOutputChannels; tc += THREADS_PER_BLOCK) {
      int i2 = (tc % displacement_size - displacement_rad) * stride2;
      int j2 = (tc / displacement_size - displacement_rad) * stride2;

      int xmin = (x - kernel_rad - max_displacement - i2) / stride1;
      int ymin = (y - kernel_rad - max_displacement - j2) / stride1;

      int xmax = (x + kernel_rad - max_displacement - i2) / stride1;
      int ymax = (y + kernel_rad - max_displacement - j2) / stride1;

      if (xmax < 0 || ymax < 0 || xmin >= outputWidth || ymin >= outputHeight) {
          // assumes gradInput2 is pre-allocated and zero filled
        continue;
      }

      if (xmin > xmax || ymin > ymax) {
          // assumes gradInput2 is pre-allocated and zero filled
          continue;
      }

      xmin = max(0,xmin);
      xmax = min(outputWidth-1,xmax);

      ymin = max(0,ymin);
      ymax = min(outputHeight-1,ymax);
      
      int indx1 = n * pdimyxc + (y - j2)* pdimxc + (x - i2) * pdimc + c;
      scalar_t val1 = rInput1[indx1];

      for (int j = ymin; j <= ymax; ++j) {
        for (int i = xmin; i <= xmax; ++i) {
          int tindx = n * tdimcyx + tc * tdimyx + j * tdimx + i;
          prod_sum[tch_off] += gradOutput[tindx] * val1;
        }
      }
    }

    __syncthreads();

    if(tch_off == 0) {
      scalar_t reduce_sum = 0;
      for(int idx = 0; idx < THREADS_PER_BLOCK; idx++) {
          reduce_sum += prod_sum[idx];
      }
      const int indx2 = n * odimcyx + c * odimyx + (y - pad_size) * odimx + (x - pad_size);
      gradInput2[indx2] = reduce_sum / nelems;
    }

}

int correlation_forward_cuda_kernel(at::Tensor& output,
                                    int ob,
                                    int oc,
                                    int oh,
                                    int ow,
                                    int osb,
                                    int osc,
                                    int osh,
                                    int osw,

                                    at::Tensor& input1,
                                    int ic,
                                    int ih,
                                    int iw,
                                    int isb,
                                    int isc,
                                    int ish,
                                    int isw,

                                    at::Tensor& input2,
                                    int gc,
                                    int gsb,
                                    int gsc,
                                    int gsh,
                                    int gsw,

                                    at::Tensor& rInput1,
                                    at::Tensor& rInput2,
                                    int pad_size,
                                    int kernel_size,
                                    int max_displacement,
                                    int stride1,
                                    int stride2,
                                    int corr_type_multiply,
                                    hipStream_t stream) 
{

   int batchSize = ob;

   int nInputChannels = ic;
   int inputWidth = iw;
   int inputHeight = ih;

   int nOutputChannels = oc;
   int outputWidth = ow;
   int outputHeight = oh;

   dim3 blocks_grid(batchSize, inputHeight, inputWidth);
   dim3 threads_block(THREADS_PER_BLOCK);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input1.type(), "channels_first_fwd_1", ([&] {

  channels_first<scalar_t><<<blocks_grid,threads_block, 0, stream>>>(
      input1.data<scalar_t>(), rInput1.data<scalar_t>(), nInputChannels, inputHeight, inputWidth, pad_size);

  }));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input2.type(), "channels_first_fwd_2", ([&] {

  channels_first<scalar_t><<<blocks_grid,threads_block, 0, stream>>> (
      input2.data<scalar_t>(), rInput2.data<scalar_t>(), nInputChannels, inputHeight, inputWidth, pad_size);

  }));

   dim3 threadsPerBlock(THREADS_PER_BLOCK);
   dim3 totalBlocksCorr(batchSize, outputHeight, outputWidth);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input1.type(), "correlation_forward", ([&] {

   correlation_forward<scalar_t><<<totalBlocksCorr, threadsPerBlock, 0, stream>>> 
                        (output.data<scalar_t>(), nOutputChannels, outputHeight, outputWidth,
                         rInput1.data<scalar_t>(), nInputChannels, inputHeight, inputWidth,
                         rInput2.data<scalar_t>(),
                         pad_size,
                         kernel_size,
                         max_displacement,
                         stride1,
                         stride2);

  }));

  hipError_t err = hipGetLastError();


  // check for errors
  if (err != hipSuccess) {
    printf("error in correlation_forward_cuda_kernel: %s\n", hipGetErrorString(err));
    return 0;
  }

  return 1;
}


int correlation_backward_cuda_kernel(
                                    at::Tensor& gradOutput,
                                    int gob,
                                    int goc,
                                    int goh,
                                    int gow,
                                    int gosb,
                                    int gosc,
                                    int gosh,
                                    int gosw,

                                    at::Tensor& input1,
                                    int ic,
                                    int ih,
                                    int iw,
                                    int isb,
                                    int isc,
                                    int ish,
                                    int isw,

                                    at::Tensor& input2,
                                    int gsb,
                                    int gsc,
                                    int gsh,
                                    int gsw,

                                    at::Tensor& gradInput1,
                                    int gisb,
                                    int gisc,
                                    int gish,
                                    int gisw,

                                    at::Tensor& gradInput2,
                                    int ggc,
                                    int ggsb,
                                    int ggsc,
                                    int ggsh,
                                    int ggsw,

                                    at::Tensor& rInput1,
                                    at::Tensor& rInput2,
                                    int pad_size,
                                    int kernel_size,
                                    int max_displacement,
                                    int stride1,
                                    int stride2,
                                    int corr_type_multiply,
                                    hipStream_t stream)
{

    int batchSize = gob;
    int num = batchSize;

    int nInputChannels = ic;
    int inputWidth = iw;
    int inputHeight = ih;

    int nOutputChannels = goc;
    int outputWidth = gow;
    int outputHeight = goh;

    dim3 blocks_grid(batchSize, inputHeight, inputWidth);
    dim3 threads_block(THREADS_PER_BLOCK);


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input1.type(), "lltm_forward_cuda", ([&] {

        channels_first<scalar_t><<<blocks_grid, threads_block, 0, stream>>>(
            input1.data<scalar_t>(),
            rInput1.data<scalar_t>(),
            nInputChannels,
            inputHeight,
            inputWidth,
            pad_size
        );
    }));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input2.type(), "lltm_forward_cuda", ([&] {

        channels_first<scalar_t><<<blocks_grid, threads_block, 0, stream>>>(
            input2.data<scalar_t>(),
            rInput2.data<scalar_t>(),
            nInputChannels,
            inputHeight,
            inputWidth,
            pad_size
        );
    }));

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 totalBlocksCorr(inputHeight, inputWidth, nInputChannels);

    for (int n = 0; n < num; ++n) {

      AT_DISPATCH_FLOATING_TYPES_AND_HALF(input2.type(), "lltm_forward_cuda", ([&] {


          correlation_backward_input1<scalar_t><<<totalBlocksCorr, threadsPerBlock, 0, stream>>> (
              n, gradInput1.data<scalar_t>(), nInputChannels, inputHeight, inputWidth,
              gradOutput.data<scalar_t>(), nOutputChannels, outputHeight, outputWidth,
              rInput2.data<scalar_t>(),
              pad_size,
              kernel_size,
              max_displacement,
              stride1,
              stride2);
      }));
    }

    for(int n = 0; n < batchSize; n++) {

      AT_DISPATCH_FLOATING_TYPES_AND_HALF(rInput1.type(), "lltm_forward_cuda", ([&] {

        correlation_backward_input2<scalar_t><<<totalBlocksCorr, threadsPerBlock, 0, stream>>>(
            n, gradInput2.data<scalar_t>(), nInputChannels, inputHeight, inputWidth,
            gradOutput.data<scalar_t>(), nOutputChannels, outputHeight, outputWidth,
            rInput1.data<scalar_t>(),
            pad_size,
            kernel_size,
            max_displacement,
            stride1,
            stride2);

        }));
    }

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in correlation_backward_cuda_kernel: %s\n", hipGetErrorString(err));
    return 0;
  }

  return 1;
}
