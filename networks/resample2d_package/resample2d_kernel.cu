#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>

#define CUDA_NUM_THREADS 512 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])

template <typename scalar_t>
__global__ void kernel_resample2d_update_output(const int n, 
                                               const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
                                               const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride, 
                                               scalar_t* __restrict__ output, const long4 output_size, const long4 output_stride, int kernel_size, bool bilinear) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    scalar_t val = 0.0f;

    int dim_b = DIM0(output_size);
    int dim_c = DIM1(output_size);
    int dim_h = DIM2(output_size);
    int dim_w = DIM3(output_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    scalar_t dx = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy = DIM3_INDEX(input2, b, 1, y, x);

    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;
    scalar_t alpha = xf - floor(xf); // alpha
    scalar_t beta = yf - floor(yf); // beta

    if (bilinear) {
        int xL = max(min( int (floor(xf)),    dim_w-1), 0);
        int xR = max(min( int (floor(xf)+1), dim_w -1), 0);
        int yT = max(min( int (floor(yf)),    dim_h-1), 0);
        int yB = max(min( int (floor(yf)+1),  dim_h-1), 0);

        for (int fy = 0; fy < kernel_size; fy += 1) {
            for (int fx = 0; fx < kernel_size; fx += 1) {
                val += static_cast<float>((1. - alpha)*(1. - beta) * DIM3_INDEX(input1, b, c, yT + fy, xL + fx));
                val += static_cast<float>((alpha)*(1. - beta) * DIM3_INDEX(input1, b, c, yT + fy, xR + fx));
                val += static_cast<float>((1. - alpha)*(beta) * DIM3_INDEX(input1, b, c, yB + fy, xL + fx));
                val += static_cast<float>((alpha)*(beta) * DIM3_INDEX(input1, b, c, yB + fy, xR + fx));
            }
        }

        output[index] = val;
    }
    else {
        int xN = max(min( int (floor(xf + 0.5)), dim_w - 1), 0);
        int yN = max(min( int (floor(yf + 0.5)), dim_h - 1), 0);

        output[index] = static_cast<float> ( DIM3_INDEX(input1, b, c, yN, xN) );
    }

}


template <typename scalar_t>
__global__ void kernel_resample2d_backward_input1(
    const int n, const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
    const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride,
    const scalar_t* __restrict__ gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride,
    scalar_t* __restrict__ gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size, bool bilinear) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    int dim_b = DIM0(gradOutput_size);
    int dim_c = DIM1(gradOutput_size);
    int dim_h = DIM2(gradOutput_size);
    int dim_w = DIM3(gradOutput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    scalar_t dx = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy = DIM3_INDEX(input2, b, 1, y, x);

    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;
    scalar_t alpha = xf - int(xf); // alpha
    scalar_t beta = yf - int(yf); // beta

    int idim_h = DIM2(input1_size);
    int idim_w = DIM3(input1_size);

    int xL = max(min( int (floor(xf)),    idim_w-1), 0);
    int xR = max(min( int (floor(xf)+1), idim_w -1), 0);
    int yT = max(min( int (floor(yf)),    idim_h-1), 0);
    int yB = max(min( int (floor(yf)+1),  idim_h-1), 0);

    for (int fy = 0; fy < kernel_size; fy += 1) {
        for (int fx = 0; fx < kernel_size; fx += 1) {
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT + fy), (xL + fx)), (1-alpha)*(1-beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT + fy), (xR + fx)),   (alpha)*(1-beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB + fy), (xL + fx)),   (1-alpha)*(beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB + fy), (xR + fx)),     (alpha)*(beta) * DIM3_INDEX(gradOutput, b, c, y, x));
        }
    }

}

template <typename scalar_t>
__global__ void kernel_resample2d_backward_input2(
    const int n, const scalar_t* __restrict__ input1, const long4 input1_size, const long4 input1_stride,
    const scalar_t* __restrict__ input2, const long4 input2_size, const long4 input2_stride,
    const scalar_t* __restrict__ gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride,
    scalar_t* __restrict__ gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size, bool bilinear) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    scalar_t output = 0.0;
    int kernel_rad = (kernel_size - 1)/2;

    int dim_b = DIM0(gradInput_size);
    int dim_c = DIM1(gradInput_size);
    int dim_h = DIM2(gradInput_size);
    int dim_w = DIM3(gradInput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    int odim_c = DIM1(gradOutput_size);

    scalar_t dx = DIM3_INDEX(input2, b, 0, y, x);
    scalar_t dy = DIM3_INDEX(input2, b, 1, y, x);

    scalar_t xf = static_cast<scalar_t>(x) + dx;
    scalar_t yf = static_cast<scalar_t>(y) + dy;

    int xL = max(min( int (floor(xf)),    dim_w-1), 0);
    int xR = max(min( int (floor(xf)+1), dim_w -1), 0);
    int yT = max(min( int (floor(yf)),    dim_h-1), 0);
    int yB = max(min( int (floor(yf)+1),  dim_h-1), 0);
    
    if (c % 2) {
        float gamma = 1 - (xf - floor(xf)); // alpha
        for (int i = 0; i <= 2*kernel_rad; ++i) {
            for (int j = 0; j <= 2*kernel_rad; ++j) {
                for (int ch = 0; ch < odim_c; ++ch) {
                    output += (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xL + i));
                    output -= (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xL + i));
                    output += (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xR + i));
                    output -= (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xR + i));
                }
            }
        }
    }
    else {
        float gamma = 1 - (yf - floor(yf)); // alpha
        for (int i = 0; i <= 2*kernel_rad; ++i) {
            for (int j = 0; j <= 2*kernel_rad; ++j) {
                for (int ch = 0; ch < odim_c; ++ch) {
                    output += (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xR + i));
                    output -= (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xL + i));
                    output += (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xR + i));
                    output -= (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xL + i));
                }
            }
        }

    }

    gradInput[index] = output;

}

void resample2d_kernel_forward(
    at::Tensor& input1, 
    at::Tensor& input2,
    at::Tensor& output, 
    int kernel_size,
    bool bilinear) {

    int n = output.numel();

    const long4 input1_size = make_long4(input1.size(0), input1.size(1), input1.size(2), input1.size(3));
    const long4 input1_stride = make_long4(input1.stride(0), input1.stride(1), input1.stride(2), input1.stride(3));

    const long4 input2_size = make_long4(input2.size(0), input2.size(1), input2.size(2), input2.size(3));
    const long4 input2_stride = make_long4(input2.stride(0), input2.stride(1), input2.stride(2), input2.stride(3));

    const long4 output_size = make_long4(output.size(0), output.size(1), output.size(2), output.size(3));
    const long4 output_stride = make_long4(output.stride(0), output.stride(1), output.stride(2), output.stride(3));

    // TODO: when atomicAdd gets resolved, change to AT_DISPATCH_FLOATING_TYPES_AND_HALF
//    AT_DISPATCH_FLOATING_TYPES(input1.type(), "resample_forward_kernel", ([&] {

        kernel_resample2d_update_output<float><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
//at::globalContext().getCurrentCUDAStream() >>>(
            n,
            input1.data<float>(),
            input1_size,
            input1_stride, 
            input2.data<float>(),
            input2_size,
            input2_stride,
            output.data<float>(),
            output_size,
            output_stride,
            kernel_size,
            bilinear);

//    }));

        // TODO: ATen-equivalent check

       //    THCudaCheck(hipGetLastError());

}

void resample2d_kernel_backward(
    at::Tensor& input1,
    at::Tensor& input2,
    at::Tensor& gradOutput,
    at::Tensor& gradInput1,
    at::Tensor& gradInput2,
    int kernel_size,
    bool bilinear) {

    int n = gradOutput.numel();

    const long4 input1_size = make_long4(input1.size(0), input1.size(1), input1.size(2), input1.size(3));
    const long4 input1_stride = make_long4(input1.stride(0), input1.stride(1), input1.stride(2), input1.stride(3));

    const long4 input2_size = make_long4(input2.size(0), input2.size(1), input2.size(2), input2.size(3));
    const long4 input2_stride = make_long4(input2.stride(0), input2.stride(1), input2.stride(2), input2.stride(3));

    const long4 gradOutput_size = make_long4(gradOutput.size(0), gradOutput.size(1), gradOutput.size(2), gradOutput.size(3));
    const long4 gradOutput_stride = make_long4(gradOutput.stride(0), gradOutput.stride(1), gradOutput.stride(2), gradOutput.stride(3));

    const long4 gradInput1_size = make_long4(gradInput1.size(0), gradInput1.size(1), gradInput1.size(2), gradInput1.size(3));
    const long4 gradInput1_stride = make_long4(gradInput1.stride(0), gradInput1.stride(1), gradInput1.stride(2), gradInput1.stride(3));

//    AT_DISPATCH_FLOATING_TYPES(input1.type(), "resample_backward_input1", ([&] {

        kernel_resample2d_backward_input1<float><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
//at::globalContext().getCurrentCUDAStream() >>>(
            n, 
            input1.data<float>(), 
            input1_size,
            input1_stride,
            input2.data<float>(),
            input2_size, 
            input2_stride,
            gradOutput.data<float>(),
            gradOutput_size,
            gradOutput_stride,
            gradInput1.data<float>(),
            gradInput1_size,
            gradInput1_stride, 
            kernel_size,
            bilinear
        );

//    }));

    const long4 gradInput2_size = make_long4(gradInput2.size(0), gradInput2.size(1), gradInput2.size(2), gradInput2.size(3));
    const long4 gradInput2_stride = make_long4(gradInput2.stride(0), gradInput2.stride(1), gradInput2.stride(2), gradInput2.stride(3));

    n = gradInput2.numel();

//    AT_DISPATCH_FLOATING_TYPES(gradInput2.type(), "resample_backward_input2", ([&] {


        kernel_resample2d_backward_input2<float><<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream() >>>(
//at::globalContext().getCurrentCUDAStream() >>>(
            n, 
            input1.data<float>(), 
            input1_size, 
            input1_stride,
            input2.data<float>(), 
            input2_size,
            input2_stride,
            gradOutput.data<float>(),
            gradOutput_size,
            gradOutput_stride,
            gradInput2.data<float>(),
            gradInput2_size,
            gradInput2_stride,
            kernel_size,
            bilinear
       );

//    }));

    // TODO: Use the ATen equivalent to get last error

    //    THCudaCheck(hipGetLastError());

}
